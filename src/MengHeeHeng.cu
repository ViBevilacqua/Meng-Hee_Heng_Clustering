#include "hip/hip_runtime.h"
#include <cfloat>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <opencv2/opencv.hpp>
#include <cmath>
#include <stdio.h>
#include "MengHeeHeng.cuh"
#include <hip/hip_runtime.h>
using namespace std;
using namespace cv;

/*

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

*/

__global__ void find2PixelGPU(unsigned char* img_b_device, unsigned char* img_g_device, unsigned char* img_r_device, float *meanPartial_device, short rows, short cols);
__global__ void updateClusterGPU(unsigned char* img_b_device, unsigned char* img_g_device, unsigned char* img_r_device, unsigned int *idCluster_device, unsigned int *meanUpdate_device, float *centroidi_device , short rows, short cols, short k);
__global__ void findFarPixelGPU(unsigned char* img_b_device, unsigned char* img_g_device, unsigned char* img_r_device, unsigned int* idCluster_device, float* centroidi_device, float* partialDist_device , short rows, short cols, short k);

void MengHeeHengGPU(unsigned char* img_host_b,
    unsigned char* img_host_g,
    unsigned char* img_host_r,
    Mat img_output,
    short rows,
    short cols){

      //unsigned char *cluster_b_device, *cluster_g_device, *cluster_r_device;
      float *meanPartial_host, *meanPartial_device;
      unsigned char *img_b_device, *img_g_device, *img_r_device;

      double maxDist = FLT_MIN;
      bool uscita;

      unsigned int size;

      dim3 num_blocks, num_threads_per_block(4,4);


      //TO-DO = inizializza num block e thread
      num_blocks.y = rows/num_threads_per_block.y+((rows%num_threads_per_block.y)==0? 0:1);
      num_blocks.x = cols/num_threads_per_block.x+((cols%num_threads_per_block.x)==0? 0:1);

      cout << "num_blocks.y: " << num_blocks.y<<endl;
      cout << "num_blocks.x: " << num_blocks.x<<endl;
      size=rows*cols*sizeof(unsigned char);
      hipMalloc((void**)&img_b_device, size);
      hipMalloc((void**)&img_g_device, size);
      hipMalloc((void**)&img_r_device, size);

      //trasferimento img da host to device
      hipMemcpy(img_b_device, img_host_b, size, hipMemcpyHostToDevice);
      hipMemcpy(img_g_device, img_host_g, size, hipMemcpyHostToDevice);
      hipMemcpy(img_r_device, img_host_r, size, hipMemcpyHostToDevice);

      //TO-DO: TROVARE I 2 PIXEL CON DISTANZA MAX
      int size_meanPartial = 5*num_blocks.x*num_blocks.y*sizeof(float);
      meanPartial_host = (float*)malloc(size_meanPartial);
      hipMalloc((void**)&meanPartial_device, size_meanPartial);

      int ShMemSize=5*num_threads_per_block.x*num_threads_per_block.y*sizeof(float);

      //1. Trovare i 2 pixel con i colori piu' distanti
      find2PixelGPU<<< num_blocks, num_threads_per_block, ShMemSize >>> (img_b_device, img_g_device, img_r_device, meanPartial_device, rows, cols);
      hipDeviceSynchronize();
    
      hipMemcpy(meanPartial_host, meanPartial_device , size_meanPartial, hipMemcpyDeviceToHost);

      float *centroidi_host;
      int k = 2;
      centroidi_host = (float*)malloc(k*3*sizeof(float));
      
      for(int i=0; i<num_blocks.x*num_blocks.y; i++){
        if (meanPartial_host[i*5+4] > maxDist){
          maxDist = meanPartial_host[i*5+4];
          int indRow = static_cast<int>(meanPartial_host[i*5]);
          int indCol = static_cast<int>(meanPartial_host[i*5+1]);
          centroidi_host[0] = static_cast<float>(img_host_b[indRow*cols+indCol]);
          centroidi_host[1] = static_cast<float>(img_host_g[indRow*cols+indCol]);
          centroidi_host[2] = static_cast<float>(img_host_r[indRow*cols+indCol]);
          indRow = static_cast<int>(meanPartial_host[i*5+2]);
          indCol = static_cast<int>(meanPartial_host[i*5+3]);
          centroidi_host[3] = static_cast<float>(img_host_b[indRow*cols+indCol]);          
          centroidi_host[4] = static_cast<float>(img_host_g[indRow*cols+indCol]);   
          centroidi_host[5] = static_cast<float>(img_host_r[indRow*cols+indCol]);
        }
      }
      //cout<<"tra poco hipFree(meanPartial_device)"<<endl;
      hipFree(meanPartial_device);
      free(meanPartial_host);

      cout << "[GPU] Le medie trovate nei primi due centroidi sono media[0]= " << centroidi_host[0]  << " media[1]= " << centroidi_host[1]  << endl;
      cout << "[GPU] La maxDist = " << maxDist << endl;

      unsigned int *meanUpdate_host, *meanUpdate_device;
      unsigned int size_meanUpdate = k*4*sizeof(unsigned int);
      meanUpdate_host = (unsigned int*)malloc(size_meanUpdate);
      hipMalloc((void**)&meanUpdate_device, size_meanUpdate);

      float *centroidi_device;
      hipMalloc((void**)&centroidi_device, k*3*sizeof(float));

      unsigned int *idCluster_host, *idCluster_device;
      float *partialDist_host, *partialDist_device; //si usa nel do-while
      unsigned int size_idCluster = rows*cols*sizeof(unsigned int);
      idCluster_host = (unsigned int *)malloc(size_idCluster);
      hipMalloc((void**)&idCluster_device, size_idCluster);

      hipMemcpy(centroidi_device, centroidi_host, k*3*sizeof(float), hipMemcpyHostToDevice);

      unsigned int ShMemSize_Update=4*k*sizeof(unsigned int);

      hipMemset(meanUpdate_device, 0, size_meanUpdate);

      //2. Clustering per prossimita', cioe' raggruppare tutti i pixel dell'immagine nel cluster con distanza minima (cluster più vicino)
      updateClusterGPU<<<num_blocks, num_threads_per_block, ShMemSize_Update>>>(img_b_device, img_g_device, img_r_device, idCluster_device, meanUpdate_device, centroidi_device, rows, cols, k); //kernel
      hipDeviceSynchronize();
      
      hipMemcpy(meanUpdate_host, meanUpdate_device , size_meanUpdate, hipMemcpyDeviceToHost);
      hipMemcpy(idCluster_host, idCluster_device , size_idCluster, hipMemcpyDeviceToHost);

      for(int i=0; i<k; i++){
        unsigned int numerator_b = meanUpdate_host[i];
        unsigned int numerator_g = meanUpdate_host[k+i];
        unsigned int numerator_r = meanUpdate_host[2*k+i];
        unsigned int denominator = meanUpdate_host[3*k+i];
 
        //cout << "[GPU] numerator_b = " << numerator_b << " numerator_g = " << numerator_g << " numerator_r = " << numerator_b << " denominator = " << denominator << endl;
        
        centroidi_host[i*3] = numerator_b / denominator; 
        centroidi_host[i*3+1] = numerator_g / denominator;
        centroidi_host[i*3+2] = numerator_r / denominator;

        //cout << "[GPU] Valori Centroidi del cluster " << i << " B: " << centroidi_host[i*3] << " G: " << centroidi_host[i*3+1] << " R: " << centroidi_host[i*3+2] << endl;
      }

     

      do{
        float d  = FLT_MIN;

        int ShMemSize_Far=4*num_threads_per_block.x*num_threads_per_block.y*sizeof(float);
        hipFree(centroidi_device);
        hipMalloc((void**)&centroidi_device, k*3*sizeof(float));
        hipMemcpy(centroidi_device, centroidi_host, k*3*sizeof(float), hipMemcpyHostToDevice);

        int sizePartialDist = 4*num_blocks.x*num_blocks.y*sizeof(float);
        partialDist_host = (float*)malloc(sizePartialDist);
        hipMalloc((void**)&partialDist_device, sizePartialDist);

        //3. Tra tutti i cluster, trovare il pixel [x] avente la massima distanza [d] dalla propria media di cluster.
        findFarPixelGPU<<<num_blocks, num_threads_per_block, ShMemSize_Far>>>(img_b_device, img_g_device, img_r_device, idCluster_device, centroidi_device, partialDist_device , rows, cols, k);
        hipDeviceSynchronize();
      
        hipMemcpy(partialDist_host, partialDist_device, sizePartialDist, hipMemcpyDeviceToHost);

        float x[3] = {0.0, 0.0, 0.0};

        for(int i=0; i<num_blocks.x*num_blocks.y; i++){
          
          if (partialDist_host[i*4+3] > d){
            d = partialDist_host[i*4+3];
            x[0] = partialDist_host[i*4];
            x[1] = partialDist_host[i*4+1];
            x[2] = partialDist_host[i*4+2];
          }
        }

        //Calcolare la distanza tra ogni coppia di cluster
        vector <double> distanzaCoppie;
        for(int i=0; i<k; i++){
          for(int j=i+1; j<k; j++){
            distanzaCoppie.push_back(sqrt(pow(centroidi_host[i*3] - centroidi_host[j*3], 2) + pow(centroidi_host[i*3+1]-centroidi_host[j*3+1], 2) + pow(centroidi_host[i*3+2]-centroidi_host[j*3+2], 2)));          
          }          
        }

        //4. Calcolare la media [q] tra tutte le distanze delle coppie di cluster
        double q = mean(distanzaCoppie)[0]; 

        //5. Calcolare la media [q] tra tutte le distanze delle coppie di cluster
        if( d > q/2){
          uscita = false;

          k++;
          ShMemSize_Update = 4*k*sizeof(unsigned int);
          centroidi_host = (float*)realloc(centroidi_host, k*3*sizeof(float));
          //Inizialmente si considera il valore del pixel [x] come rappresentante del nuovo cluster
          centroidi_host[(k-1)*3] = x[0];
          centroidi_host[(k-1)*3+1] = x[1];
          centroidi_host[(k-1)*3+2] = x[2];

          hipFree(centroidi_device);
          hipMalloc((void**)&centroidi_device, k*3*sizeof(float));
          hipMemcpy(centroidi_device, centroidi_host, k*3*sizeof(float), hipMemcpyHostToDevice);

          hipFree(meanUpdate_device);
          meanUpdate_host = (unsigned int*)realloc(meanUpdate_host, ShMemSize_Update);
          hipMalloc((void**)&meanUpdate_device, ShMemSize_Update);

          hipMemset(meanUpdate_device, 0, ShMemSize_Update);

          
          ////Essendoci un nuovo cluster, per ogni pixel si ricalcola il cluster con distanza minima.
          updateClusterGPU<<<num_blocks, num_threads_per_block, ShMemSize_Update>>>(img_b_device, img_g_device, img_r_device, idCluster_device, meanUpdate_device, centroidi_device, rows, cols, k);
          hipDeviceSynchronize();
          
          hipMemcpy(meanUpdate_host, meanUpdate_device , ShMemSize_Update, hipMemcpyDeviceToHost);
          hipMemcpy(idCluster_host, idCluster_device , size_idCluster, hipMemcpyDeviceToHost);

          for(int i=0; i<k; i++){
            unsigned int numerator_b = meanUpdate_host[i];
            unsigned int numerator_g = meanUpdate_host[k+i];
            unsigned int numerator_r = meanUpdate_host[2*k+i];
            unsigned int denominator = meanUpdate_host[3*k+i];
            
            centroidi_host[i*3] = numerator_b / denominator; 
            centroidi_host[i*3+1] = numerator_g / denominator;
            centroidi_host[i*3+2] = numerator_r / denominator;
          }

        }else
          uscita = true;
      //Se non e' stato creato un nuovo cluster allora termina la fase di costruzione dei cluster, altrimenti si torna a controllare l'ipotetica presenza di nuovi cluster
      }while(!uscita);

      //Quando i cluster sono stati costruiti, l'algoritmo di Mang-Heng Hee puo' essere considerato concluso, l'ultima cosa da fare e' salvare i cluster appena costruiti in una Mat di output,
      //Inserendo come valore di ogni pixel media del corrispettivo cluster con la rappresentazione di colore RGB.
      for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
          int ind_k = idCluster_host[i*cols+j];
          img_output.at<Vec3b>(i,j) = Vec3b( (unsigned char)centroidi_host[ind_k*3], (unsigned char)centroidi_host[ind_k*3+1], (unsigned char)centroidi_host[ind_k*3+2]);
        }
      }

      cout << "[GPU] Numero di Cluster trovati = " << k << endl;

      free(centroidi_host);
      free(idCluster_host);
      free(meanUpdate_host);
      free(partialDist_host);
      hipFree(img_b_device);
      hipFree(img_g_device);
      hipFree(img_r_device);
      hipFree(idCluster_device);
      hipFree(centroidi_device);
      hipFree(meanUpdate_device);
      hipFree(partialDist_device);           
    }

    __global__ void findFarPixelGPU(unsigned char* img_b_device, unsigned char* img_g_device, unsigned char* img_r_device, unsigned int* idCluster_device, float* centroidi_device, float* partialDist_device , short rows, short cols, short k){
        extern __shared__ float sm_far[];

        int indexRow=threadIdx.y + blockIdx.y*blockDim.y;
        int indexCol=threadIdx.x + blockIdx.x*blockDim.x;

        if(indexRow<rows && indexCol<cols){
          
          int id = idCluster_device[indexRow*cols+indexCol];
          float b_val = (float)img_b_device[indexRow*cols+indexCol];
          float g_val = (float)img_g_device[indexRow*cols+indexCol];
          float r_val = (float)img_r_device[indexRow*cols+indexCol];

          float distanza = sqrt(pow(b_val-centroidi_device[id*3], 2) + pow(g_val-centroidi_device[id*3+1], 2) + pow(r_val-centroidi_device[id*3+2], 2));
          sm_far[(4*threadIdx.x)+(threadIdx.y*blockDim.x*4)] = b_val;
          sm_far[((4*threadIdx.x)+(threadIdx.y*blockDim.x*4))+1] = g_val;
          sm_far[((4*threadIdx.x)+(threadIdx.y*blockDim.x*4))+2] = r_val;
          sm_far[((4*threadIdx.x)+(threadIdx.y*blockDim.x*4))+3] = distanza;
          __syncthreads();

          if (threadIdx.x==0 && threadIdx.y==0) {

            float maxDist = FLT_MIN;

            for(int i=0; i<blockDim.y; i++){
              for (int j = 0; j < blockDim.x; j++) {          
                if( (i+blockIdx.y*blockDim.y)<rows && (j+blockIdx.x*blockDim.x)<cols && sm_far[((4*j)+(i*blockDim.x*4))+3] > maxDist){
                  maxDist = sm_far[((4*j)+(i*blockDim.x*4))+3];
                  b_val = sm_far[((4*j)+(i*blockDim.x*4))];
                  g_val = sm_far[((4*j)+(i*blockDim.x*4))+1];
                  r_val = sm_far[((4*j)+(i*blockDim.x*4))+2];                        
                }  
              }
            }

            partialDist_device[(4*blockIdx.x)+(blockIdx.y*gridDim.x*4)] = b_val;
            partialDist_device[((4*blockIdx.x)+(blockIdx.y*gridDim.x*4))+1] = g_val;
            partialDist_device[((4*blockIdx.x)+(blockIdx.y*gridDim.x*4))+2] = r_val;
            partialDist_device[((4*blockIdx.x)+(blockIdx.y*gridDim.x*4))+3] = maxDist;
          }   
        }
    }


    __global__ void updateClusterGPU(unsigned char* img_b_device, unsigned char* img_g_device, unsigned char* img_r_device, unsigned int *idCluster_device, unsigned int *meanUpdate_device, float *centroidi_device,  short rows, short cols, short k){

        extern __shared__ unsigned int sm_up[];

        int indexRow=threadIdx.y + blockIdx.y*blockDim.y;
        int indexCol=threadIdx.x + blockIdx.x*blockDim.x;

        if (threadIdx.x==0 && threadIdx.y==0) {
          for(int i = 0; i< k*4; i++){
            sm_up[i] = 0;
          }
        }


        if(indexRow<rows && indexCol<cols){

          float minDist = FLT_MAX;
          int idCluster;

          for(int i = 0; i<k; i++){
            float distanza = sqrt(pow((float)img_b_device[indexRow*cols+indexCol]-centroidi_device[i*3], 2) + pow((float)img_g_device[indexRow*cols+indexCol]-centroidi_device[i*3+1], 2) + pow((float)img_r_device[indexRow*cols+indexCol]-centroidi_device[i*3+2], 2));
            if(distanza < minDist){
              idCluster = i;
              minDist = distanza;
            }
          }

          idCluster_device[indexRow*cols+indexCol] = idCluster;

          __syncthreads();

          atomicAdd(&(sm_up[idCluster]), img_b_device[indexRow*cols+indexCol]);
          atomicAdd(&(sm_up[idCluster+1*k]), img_g_device[indexRow*cols+indexCol]);
          atomicAdd(&(sm_up[idCluster+2*k]), img_r_device[indexRow*cols+indexCol]);
          atomicAdd(&(sm_up[idCluster+3*k]), 1);
          
          __syncthreads();

          if (threadIdx.x==0 && threadIdx.y==0) {
            for(int j=0; j<k; j++){
              atomicAdd(&meanUpdate_device[j], sm_up[j]);
              atomicAdd(&meanUpdate_device[k+j], sm_up[k+j]);
              atomicAdd(&meanUpdate_device[2*k+j], sm_up[2*k+j]);
              atomicAdd(&meanUpdate_device[3*k+j], sm_up[3*k+j]);
            }        
          }    
       }
    }


    __global__ void find2PixelGPU(unsigned char* img_b_device, unsigned char* img_g_device, unsigned char* img_r_device, float *meanPartial_device, short rows, short cols){

        extern __shared__ float sm[];

        int indexRow=threadIdx.y + blockIdx.y*blockDim.y;
        int indexCol=threadIdx.x + blockIdx.x*blockDim.x;
        float maxDist = FLT_MIN;
        //if(indexRow == 384 && indexCol==214)
        //printf("blockIdx.x: %d, blockIdx.y: %d \t theadID.x: %d, threadID.y: %d \t indexRow: %d, indexCol: %d\n", blockIdx.x,blockIdx.y, threadIdx.x,threadIdx.y,indexRow,indexCol);           
        if(indexRow<rows && indexCol<cols){
                  //if(blockIdx.x == 56 && blockIdx.y == 56){
                 //   printf("theadID.x: %d, threadID.y: %d\n", threadIdx.x,threadIdx.y);
                //  }
            for(int k = indexRow; k<rows; k++){
              for(int w = indexCol+1; w<cols; w++){
                float distanza = sqrt(pow((float)img_b_device[indexRow*cols+indexCol]-img_b_device[k*cols+w], 2) + pow((float)img_g_device[indexRow*cols+indexCol]-img_g_device[k*cols+w], 2) + pow((float)img_r_device[indexRow*cols+indexCol]-img_r_device[k*cols+w], 2));
                if( distanza > maxDist )
                {
                    maxDist = distanza;
                    sm[(5*threadIdx.x)+(threadIdx.y*blockDim.x*5)]= (float)indexRow; //indicizzazione per accedere alla sm ogni thread deve salvare i suoi 5 risultati parziali
                    sm[((5*threadIdx.x)+(threadIdx.y*blockDim.x*5))+1] = (float)indexCol;
                    sm[((5*threadIdx.x)+(threadIdx.y*blockDim.x*5))+2] = (float)k;
                    sm[((5*threadIdx.x)+(threadIdx.y*blockDim.x*5))+3] = (float)w;
                    sm[((5*threadIdx.x)+(threadIdx.y*blockDim.x*5))+4] = maxDist;
                }
              }
            }

        }
        __syncthreads();

        if (threadIdx.x==0 && threadIdx.y==0) {
          maxDist = FLT_MIN;
          int max_i = 0;
          int max_j = 0;
          for(int i=0; i<blockDim.y; i++){
              for (int j = 0; j < blockDim.x; j++) {
                if(sm[((5*j)+(i*blockDim.x*5))+4] > maxDist){
                  maxDist = sm[((5*j)+(i*blockDim.x*5))+4];
                  max_j = j;
                  max_i = i;
                }
              }
          }

          //(i+blockIdx.y*blockDim.y)<rows && (j+blockIdx.x*blockDim.x)<cols && 
          if(maxDist != FLT_MIN){
            meanPartial_device[(5*blockIdx.x)+(blockIdx.y*gridDim.x*5)] = sm[((5*max_j)+(max_i*blockDim.x*5))]; //indicizzazione per accedere ai migliori 5 elementi che ogni blocco deve salvarsi
            meanPartial_device[(5*blockIdx.x)+(blockIdx.y*gridDim.x*5)+1] = sm[((5*max_j)+(max_i*blockDim.x*5))+1];
            meanPartial_device[(5*blockIdx.x)+(blockIdx.y*gridDim.x*5)+2] = sm[((5*max_j)+(max_i*blockDim.x*5))+2];
            meanPartial_device[(5*blockIdx.x)+(blockIdx.y*gridDim.x*5)+3] = sm[((5*max_j)+(max_i*blockDim.x*5))+3];
            meanPartial_device[(5*blockIdx.x)+(blockIdx.y*gridDim.x*5)+4] = sm[((5*max_j)+(max_i*blockDim.x*5))+4];
          }else{
            meanPartial_device[(5*blockIdx.x)+(blockIdx.y*gridDim.x*5)+4] = FLT_MIN;
          }
        }
    }

